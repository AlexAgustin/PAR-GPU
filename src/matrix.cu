#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "../include/matrix.cuh"
#include "matrix.h"

#define THR_PER_BLOCK 1024 

__global__ void matrix_mul_add_kernel(double *c, double *a, double *b, int a_rows, int a_cols, int b_cols, double *d) {
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    double sum = 0.0;
    if (row < a_rows && col < b_cols) {
        for (int i = 0; i < a_cols; i++) {
            sum += *m_elem(a, a_cols, row, i) * *m_elem(b, b_cols, i, col);
        }
        *m_elem(c, b_cols, row, col) = sum + *m_elem(d, b_cols, row, col);
    }
}

__global__ void matrix_func_kernel(double *n, double *m, int rows, int cols, double (*func)(double)) {
    int col = threadIdx.x + blockIdx.x * blockDim.x;
    int row = threadIdx.y + blockIdx.y * blockDim.y;
    if (col < cols && row < rows) {
        *m_elem(n, cols, row, col) = func(*m_elem(m, cols, row, col));
    }
}

__global__ void matrix_mul_cnt_kernel(double *m, int rows, int cols, double cnt) {
    int col = threadIdx.x + blockIdx.x * blockDim.x;
    int row = threadIdx.y + blockIdx.y * blockDim.y;
    if (col < cols && row < rows) {
        *m_elem(m, cols, row, col) *= cnt;
    }
}

__global__ void matrix_sub_kernel(double *c, double *a, double *b, int rows, int cols){

    int col = threadIdx.x + blockIdx.x * blockDim.x;
    int row = threadIdx.y + blockIdx.y * blockDim.y;

    if (col < cols && row < rows) {
        double sum;
        sum = *m_elem(a, cols, row, col) - *m_elem(b, cols, row, col);
        *m_elem(c, cols, row, col) = sum;
    }
}

__global__ void matrix_zero_kernel(double *m, int rows, int cols){

    int col = threadIdx.x + blockIdx.x * blockDim.x;
    int row = threadIdx.y + blockIdx.y * blockDim.y;

    if (col < cols && row < rows) {
        *m_elem(m, cols, row, col) = 0.0;
    }
}

__global__ void matrix_mul_dot_kernel(double *c, double *a, double *b, int rows, int cols){

    int col = threadIdx.x + blockIdx.x * blockDim.x;
    int row = threadIdx.y + blockIdx.y * blockDim.y;

    if (col < cols && row < rows) {
        double prod;
        prod = *m_elem(a, cols, row, col) * *m_elem(b, cols, row, col);
        *m_elem(c, cols, row, col) = prod;
    }
}

__global__ void matrix_transpose_kernel(double *m, double *m_t, int rows, int cols) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < rows && j < cols) {
        *m_elem(m_t, rows, j, i) = *m_elem(m, cols, i, j);
    }
}

__global__ void matrix_mul_kernel(double *c, double *a, double *b, int a_rows, int a_cols, int b_cols) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < a_rows && col < b_cols) {
        double sum = 0.0;
        for (int i = 0; i < a_cols; i++) {
            sum += *m_elem(a, a_cols, row, i) * *m_elem(b, b_cols, i, col);
        }
        *m_elem(c, b_cols, row, col) = sum;
    }
}

//------------------------------------------------------------------------------------------------------------------------------//

//Combierte los indices 2D a 1D para el acceso
double m_elem(double *m, int length, int x, int y){
    return m[length * x + y];
}

// C= A * B + D
void gpu_matrix_mul_add(double *c, double *a, double *b, int a_rows, int a_cols, int b_rows, int b_cols, double *d) {
    assert(a_cols == b_rows);
    thr_per_blk = THR_PER_BLOCK;
    blk_in_grid = ceil( (float)col / thr_per_blk );
    matrix_mul_add_kernel<<<blk_in_grid, thr_per_blk>>>(c, a, b, a_rows, a_cols, b_cols, d);
    //TODO hay que traer de vuelta los resultados de mul add
}

//Hacer uso de func a cada elem
void gpu_matrix_func(double *n, double *m, int rows, int cols, double (*func)(double)) {
    thr_per_blk = THR_PER_BLOCK;
    blk_in_grid = ceil( (float)col / thr_per_blk );
    matrix_func_kernel<<<blk_in_grid, thr_per_blk>>>(n, m, rows, cols, func);
    //TODO hay que traer de vuelta los resultados de func
}

// M * cnt
void gpu_matrix_mul_cnt(double *m, int rows, int cols, double cnt) {
    int thr_per_block = THR_PER_BLOCK;
    int blk_in_grid = ceil((float)cols / thr_per_block);
    matrix_mul_cnt_kernel<<<blk_in_grid, thr_per_block>>>(m, rows, cols, cnt);
    //TODO hay que traer de vuelta los resultados de mul cnt
}

// C = A - B
void gpu_matrix_sub(double *c, double *a, double *b, int rows, int cols){
    int thr_per_block = THR_PER_BLOCK;
    int blk_in_grid = ceil((float)cols / thr_per_block);
    matrix_sub_kernel<<<blk_in_grid, thr_per_block>>>(c, a, b, rows, cols);
    //TODO hay que traer de vuelta los resultados de sub
}

void gpu_matrix_zero(double *m, int rows, int cols){
    int thr_per_block = THR_PER_BLOCK;
    int blk_in_grid = ceil((float)cols / thr_per_block);
    matrix_zero_kernel<<<blk_in_grid, thr_per_block>>>(m, rows, cols);
    //TODO hay que traer de vuelta los resultados de sub
}

void gpu_matrix_mul_dot(double *c, double *a, double *b, int rows, int cols){
    int thr_per_block = THR_PER_BLOCK;
    int blk_in_grid = ceil((float)cols / thr_per_block);
    matrix_mul_dot_kernel<<<blk_in_grid, thr_per_block>>>(c, a, b, rows, cols);
    //TODO hay que traer de vuelta los resultados de sub
}

double *gpu_matrix_transpose(double *m, int rows, int cols) {
    double *m_t, *new_m_t;
    int i, j;

    gpuErrchk(hipMalloc(&dev_m_t, rows * cols * sizeof(double)));

    int thr_per_block = THR_PER_BLOCK;
    int blk_in_grid = ceil((float)cols / thr_per_block);

    matrix_transpose_kernel<<<blk_in_grid, thr_per_block>>>(m, dev_m_t, rows, cols);

    gpuErrchk(hipMemcpy(m_t, dev_m_t, rows * cols * sizeof(double), hipMemcpyDeviceToHost));

    gpuErrchk(hipFree(dev_m_t));

    return m_t;
}

void gpu_matrix_mul(double *c, double *a, double *b, int a_rows, int a_cols, int b_rows, int b_cols){
    int thr_per_block = THR_PER_BLOCK;
    int blk_in_grid = ceil((float)cols / thr_per_block);
    matrix_mul_kernel<<<blk_in_grid, thr_per_block>>>(c, a, b, a_rows, a_cols, b_rows, b_cols);
}

void gpu_matrix_free(double *m){

    if (m != NULL)
        gpuErrchk(hipFree(m));
}

void gpu_matrix_sum(double *c, double *a, double *b, int rows, int cols){
    int thr_per_block = THR_PER_BLOCK;
    int blk_in_grid = ceil((float)cols / thr_per_block);
    matrix_sum_kernel<<<blk_in_grid, thr_per_block>>>(c, a, b, rows, cols);
}
